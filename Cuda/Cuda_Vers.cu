#include "hip/hip_runtime.h"
// Copyright 2021 Andrey Myshkin
// Cuda Version -----

//#ifdef __INTELLISENSE___
//// in here put whatever is your favorite flavor of intellisense workarounds
//#endif
#include "hip/hip_runtime.h"
#include ""
#include "thrust\reduce.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>

// #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>


__shared__ float sharedBufMin[2048];
__shared__ float sharedBufMax[2048];

__device__ float Function(float x) {
	return (5 * sin(4 * x) - cos(x * x) * sin(x * exp(4)));
}

//__device__ void Function_New(float x, float &value) {
//	value = (5 * sin(4 * x) - cos(x * x) * sin(x * exp(4)));
//	return;
//}

__global__ void resultFunction(float* buffer, float start, float step, int size) {
	int idx = blockIdx.x;
	int dimx = blockDim.x;
	int bias = dimx * idx + threadIdx.x;
	if (bias > size - 1) return;

	float x = start + bias * step;
	buffer[bias] = Function(x);
	return;
}

__global__ void FindMinnMax(float* buffer, float* bufOutPutMin, float* bufOutPutMax) {
	extern __shared__ float sharedBufMin[];
	extern __shared__ float sharedBufMax[];

	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x;
	unsigned int dimx = blockDim.x;
	unsigned int cur = idx * dimx + threadIdx.x;

	sharedBufMin[tid] = buffer[cur];
	sharedBufMax[tid] = buffer[cur];
	__syncthreads();

	for (unsigned int i = 1; i < dimx; i *= 2) {
		int num = 2 * i * tid;
		if (num < dimx) {
			__syncthreads();
			if (threadIdx.x == 0) acquire_semaphore(&sem);
            __syncthreads();
			// Start critical section			
			if (sharedBufMin[tid] > sharedBufMin[tid + i]) sharedBufMin[tid] = sharedBufMin[tid + i];
			if (sharedBufMax[tid] < sharedBufMax[tid + i]) sharedBufMax[tid] = sharedBufMax[tid + i];
			// End critical section
			__threadfence();
			__syncthreads();
            if (threadIdx.x == 0) release_semaphore(&sem);
			__syncthreads();
		}
		__syncthreads();
	}
	if (tid == 0) {
		bufOutPutMin[idx] = sharedBufMin[0];
		bufOutPutMax[idx] = sharedBufMax[0];
	}
	return;
}


int main() {
	float start = 0;
	float end = 128;  // 2^7
	float step = 0.0625;  // 2^(-4)
	float sizeBlock = 256;
	hipError_t status;

	int size = (end - start) / step;  // 2048
	// int sizeGrid = ceil(size / sizeBlock);  // 8

	// float min, max;
	float* min = (float*)malloc(sizeof(float) * size);
	float* max = (float*)malloc(sizeof(float) * size);

	float* bufferValuesGPU = nullptr;
	// bufferValues = (float*)malloc(sizeof(float) * size);
	status = hipMalloc((void**)&bufferValuesGPU, sizeof(float) * size);
	if (status != hipSuccess) printf("Error! Incorrect allocate memory \"bufferValuesGPU\"..");

	int sizeGrid = ceil(size / sizeBlock);

	resultFunction<<<sizeGrid, sizeBlock>>>(bufferValuesGPU, start, step, size);  // ���������� ������� ���������� � �������� ������
	hipDeviceSynchronize();

	float* bufferValuesMin = nullptr;
	float* bufferValuesMax = nullptr;
	status = hipMalloc((void**)&bufferValuesMin, sizeof(float) * size);
	if (status != hipSuccess) printf("Error! Incorrect allocate memory \"bufferValuesMin\" ..");
	status = hipMalloc((void**)&bufferValuesMax, sizeof(float) * size);
	if (status != hipSuccess) printf("Error! Incorrect allocate memory \"bufferValuesMax\" ..");

	FindMinnMax<<<sizeGrid, sizeBlock>>>(bufferValuesGPU, bufferValuesMin, bufferValuesMax);  // ���������� �������� � ���������

	hipEvent_t syncEvent;
	
	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	status = hipMemcpy(min, bufferValuesMin, sizeof(float) * size, hipMemcpyDeviceToHost);
	if (status != hipSuccess) printf("Error! Incorrect hipMemcpy \"bufferValuesMin\" ..");
	status = hipMemcpy(max, bufferValuesMax, sizeof(float) * size, hipMemcpyDeviceToHost);
	if (status != hipSuccess) printf("Error! Incorrect hipMemcpy \"bufferValuesMax\" ..");

	float minGl = min[0];
	float maxGl = max[0];
	for (int i = 1; i < sizeGrid; i++) {
		if (minGl > min[i]) minGl = min[i];
		if (maxGl < max[i]) maxGl = max[i];		
	}

    printf("Function's Minimum = %lf", minGl);
	printf("Function's Maximum = &lf", maxGl);
	// printf("Function's Minimum = %lf", min[0]);
	// printf("Function's Maximum = &lf", max[0]);

	hipEventDestroy(syncEvent);

	hipFree(bufferValuesGPU);
	hipFree(bufferValuesMin);
	hipFree(bufferValuesMax);

	if (min) { free(min); min = nullptr; }
	if (max) { free(min); min = nullptr; }

	return 0;
}
